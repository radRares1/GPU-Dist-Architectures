#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <sys/timeb.h>
#include <hip/hip_runtime.h>
#include "util.h"
#include ""

__device__ void set_pixel_device(unsigned char* image, int width, int x, int y, unsigned char* c) {
    image[4 * width * y + 4 * x + 0] = c[0];
    image[4 * width * y + 4 * x + 1] = c[1];
    image[4 * width * y + 4 * x + 2] = c[2];
    image[4 * width * y + 4 * x + 3] = 255;
}


//VERY BAD DON'T DO THIS
__device__ unsigned char color2byte_device(float v) {
    float c = v * 255;
    if (c < 0) {
        c = 0;
    }
    if (c > 255) {
        c = 255;
    }
    return (unsigned char)c;
}

__device__ void hsv2rgb_device(float h, float s, float v, unsigned char* rgb)
{
    int i;
    float f, p, q, t, r, g, b;

    if (s == 0) {
        r = g = b = v;
        return;
    }

    h /= 60;
    i = (int)floor(h);
    f = h - i;
    p = v * (1 - s);
    q = v * (1 - s * f);
    t = v * (1 - s * (1 - f));

    switch (i) {
    case 0:
        r = v;
        g = t;
        b = p;
        break;
    case 1:
        r = q;
        g = v;
        b = p;
        break;
    case 2:
        r = p;
        g = v;
        b = t;
        break;
    case 3:
        r = p;
        g = q;
        b = v;
        break;
    case 4:
        r = t;
        g = p;
        b = v;
        break;
    default:
        r = v;
        g = p;
        b = q;
        break;
    }

    rgb[0] = color2byte_device(r);
    rgb[1] = color2byte_device(g);
    rgb[2] = color2byte_device(b);
}


__device__ void init_colormap_device(int len, unsigned char* map) {
    int i;
    for (i = 0; i < len; i++) {
        hsv2rgb_device(i / 4.0f, 1.0f, i / (i + 8.0f), &map[i * 3]);
    }
    map[3 * len + 0] = 0;
    map[3 * len + 1] = 0;
    map[3 * len + 2] = 0;

}


/* This should be conveted into a GPU kernel */
__global__ void generate_image(unsigned char* image, unsigned char* colormap, int width, int height, int max) {
    int row, col, index, iteration;
    double c_re, c_im, x, y, x_new;

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    int id = idy * width + idx;

    if (idx >= width || idy >= height) {
        return;
    }
    c_re = (idx - width / 2.0) * 4.0 / width;
    c_im = (idy - height / 2.0) * 4.0 / width;
    x = 0, y = 0;
    iteration = 0;
    while (x * x + y * y <= 4 && iteration < max) {
        x_new = x * x - y * y + c_re;
        y = 2 * x * y + c_im;
        x = x_new;
        iteration++;
    }
    if (iteration > max) {
        iteration = max;
    }
    set_pixel_device(image, width, idx, idy, &colormap[iteration * 3]);
}


//logicly bad why would you want to do this but it is what is is
//__global__ void generate_image_shared(unsigned char* image, int width, int height, int max) {
//    int row, col, index, iteration;
//    double c_re, c_im, x, y, x_new;
//
//    
//    ////declare it as shared
//    __shared__ unsigned char colormap[(MAX_ITERATION + 1) * 3];
//
//     // ? ? ? ? NOT OK
//    ////init on device side
//    init_colormap_device(max, colormap);
//
//    //make sure that the threads are synched after colormap init and reached the barrier
//    __syncthreads();
//
//    int idx = blockIdx.x * blockDim.x + threadIdx.x;
//    int idy = blockIdx.y * blockDim.y + threadIdx.y;
//    int id = idy * width + idx;
//
//    if (idx >= width || idy >= height) {
//        return;
//    }
//
//
//    c_re = (idx - width / 2.0) * 4.0 / width;
//    c_im = (idy - height / 2.0) * 4.0 / width;
//    x = 0, y = 0;
//    iteration = 0;
//    while (x * x + y * y <= 4 && iteration < max) {
//        x_new = x * x - y * y + c_re;
//        y = 2 * x * y + c_im;
//        x = x_new;
//        iteration++;
//    }
//    if (iteration > max) {
//        iteration = max;
//    }
//    set_pixel_device(image, width, idx, idy, &colormap[iteration * 3]);
//}

//declare colormap on constant mem
__constant__ unsigned char colormap_on_constant[(MAX_ITERATION + 1) * 3];

__global__ void generate_image_constant(unsigned char* image, int width, int height, int max) {
    int row, col, index, iteration;
    double c_re, c_im, x, y, x_new;

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    int id = idy * width + idx;

    if (idx >= width || idy >= height) {
        return;
    }


    c_re = (idx - width / 2.0) * 4.0 / width;
    c_im = (idy - height / 2.0) * 4.0 / width;
    x = 0, y = 0;
    iteration = 0;
    while (x * x + y * y <= 4 && iteration < max) {
        x_new = x * x - y * y + c_re;
        y = 2 * x * y + c_im;
        x = x_new;
        iteration++;
    }
    if (iteration > max) {
        iteration = max;
    }
    set_pixel_device(image, width, idx, idy, &colormap_on_constant[iteration * 3]);
}

int main(int argc, char** argv) {
    double times[REPEAT];
    struct timeb start, end;
    int i, r;
    char path[255];

    unsigned char* colormap = (unsigned char*)malloc((MAX_ITERATION + 1) * 3);
    unsigned char* image = (unsigned char*)malloc(WIDTH * HEIGHT * 4);

    unsigned char* device_colormap;
    unsigned char* device_image;

    hipMalloc(&device_colormap, (MAX_ITERATION + 1) * 3);
    hipMalloc(&device_image, WIDTH * HEIGHT * 4);

    init_colormap(MAX_ITERATION, colormap);

    //only for global memory kernel
    /*hipMemcpy(device_colormap, colormap, (MAX_ITERATION + 1) * 3, hipMemcpyHostToDevice);*/

    //copy it to the constant memory
    hipMemcpyToSymbol(HIP_SYMBOL(colormap_on_constant), colormap, (MAX_ITERATION + 1) * 3);

    int BLOCK_SIZE = 16;

    dim3 block(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid((WIDTH + BLOCK_SIZE - 1) / BLOCK_SIZE, (HEIGHT + BLOCK_SIZE - 1) / BLOCK_SIZE);

    for (r = 0; r < REPEAT; r++) {
        hipMemset(device_image, 0, WIDTH * HEIGHT * 4);

        ftime(&start);

        /* BEGIN: GPU implementation */

        ////kernel with normal colormap transfered to device
        //generate_image << <grid, block >> > (device_image, device_colormap, WIDTH, HEIGHT, MAX_ITERATION);

        // kernel with color map held in shared memory
        /*  generate_image_shared << <grid, block >> > (device_image, WIDTH, HEIGHT, MAX_ITERATION);
        hipDeviceSynchronize();*/

        generate_image_constant << <grid, block >> > (device_image, WIDTH, HEIGHT, MAX_ITERATION);


        /* END: GPU implementation */

        ftime(&end);
        times[r] = end.time - start.time + ((double)end.millitm - (double)start.millitm) / 1000.0;

        hipMemcpy(image, device_image, WIDTH * HEIGHT * 4, hipMemcpyDeviceToHost);
        sprintf(path, IMAGE, "gpu", r);
        save_image(path, image, WIDTH, HEIGHT);
        progress("gpu", r, times[r]);
    }
    report("gpu", times);

    hipFree(device_colormap);
    hipFree(device_image);
    free(image);
    free(colormap);
    return 0;

}